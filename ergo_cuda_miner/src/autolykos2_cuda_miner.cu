#include "hip/hip_runtime.h"
// autolykos2_cuda_miner.cu

#include "autolykos2_cuda_miner.h"
#include "blake2b_cuda.cuh"
#include <hip/hip_runtime.h>
#include <>
#include <stdint.h>
#include <stdio.h>
#include <string.h>
#include <gmp.h>
#include <string>

// Reference blake2b_sigma table defined in blake2b_cuda.cu
extern __constant__ uint8_t blake2b_sigma[12][16];

#define AUTOLYKOS2_N 26
#define AUTOLYKOS2_K 32
#define AUTOLYKOS2_M (1 << AUTOLYKOS2_N)
#define BLOCK_SIZE 256
#define GRID_SIZE 1024
#define NONCES_PER_ITER (BLOCK_SIZE * GRID_SIZE)
#define NUM_SIZE_32 8
#define K_LEN 64

#define B2B_IV(h) \
    do { \
        ((uint64_t *)(h))[0] = 0x6A09E667F2BDC928ULL; \
        ((uint64_t *)(h))[1] = 0xBB67AE8584CAA73BULL; \
        ((uint64_t *)(h))[2] = 0x3C6EF372FE94F82BULL; \
        ((uint64_t *)(h))[3] = 0xA54FF53A5F1D36F1ULL; \
        ((uint64_t *)(h))[4] = 0x510E527FADE682D1ULL; \
        ((uint64_t *)(h))[5] = 0x9B05688C2B3E6C1FULL; \
        ((uint64_t *)(h))[6] = 0x1F83D9ABFB41BD6BULL; \
        ((uint64_t *)(h))[7] = 0x5BE0CD19137E2179ULL; \
    } while(0)

__device__ __forceinline__ uint32_t cuda_swab32(uint32_t x) {
    return __byte_perm(x, x, 0x0123);
}

__device__ __forceinline__ uint64_t devROTR64(uint64_t b, int offset) {
    return (b >> offset) | (b << (64 - offset));
}

__device__ __forceinline__
void devB2B_MIX(uint64_t* v, uint64_t* m) {
    for (int r = 0; r < 12; ++r) {
        #define G(a,b,c,d,x,y) \
            v[a] = v[a] + v[b] + m[blake2b_sigma[r][x]]; \
            v[d] = devROTR64(v[d] ^ v[a], 32); \
            v[c] = v[c] + v[d]; \
            v[b] = devROTR64(v[b] ^ v[c], 24); \
            v[a] = v[a] + v[b] + m[blake2b_sigma[r][y]]; \
            v[d] = devROTR64(v[d] ^ v[a], 16); \
            v[c] = v[c] + v[d]; \
            v[b] = devROTR64(v[b] ^ v[c], 63);

        const uint8_t* s = blake2b_sigma[r];
        G(0,4,8,12,0,1);
        G(1,5,9,13,2,3);
        G(2,6,10,14,4,5);
        G(3,7,11,15,6,7);
        G(0,5,10,15,8,9);
        G(1,6,11,12,10,11);
        G(2,7,8,13,12,13);
        G(3,4,9,14,14,15);
        #undef G
    }
}

const __constant__ uint64_t ivals[8] = {
    0x6A09E667F2BDC928,
    0xBB67AE8584CAA73B,
    0x3C6EF372FE94F82B,
    0xA54FF53A5F1D36F1,
    0x510E527FADE682D1,
    0x9B05688C2B3E6C1F,
    0x1F83D9ABFB41BD6B,
    0x5BE0CD19137E2179
};

__constant__ uint8_t bound_[32];

void decodeTarget(const std::string& targetStr, uint8_t* targetBytes) {
    mpz_t targetInt;
    mpz_init_set_str(targetInt, targetStr.c_str(), 10);
    uint8_t be_bytes[32] = {0};
    size_t count = 0;
    mpz_export(be_bytes, &count, 1, 1, 0, 0, targetInt);
    for (size_t i = 0; i < 32; ++i) {
        targetBytes[i] = be_bytes[31 - i];
    }
    mpz_clear(targetInt);
}

void cpyBSymbol(const uint8_t *bound) {
    hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(bound_), bound, 32);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error in cpyBSymbol: %s\n", hipGetErrorString(err));
    }
}

__global__ void autolykos2_mining_kernel(
    const uint32_t* dataset,
    const uint8_t* header,
    uint64_t start_nonce,
    uint32_t target_hi,
    uint64_t* d_found_nonce_param,
    uint8_t* d_found_hash_param,
    bool* d_found_flag_param
) {
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t aux[32] = { 0 };
    uint32_t ind[K_LEN] = { 0 };
    uint32_t r[NUM_SIZE_32 + 1] = { 0 };
    uint8_t j = 0;

    if (tid < NONCES_PER_ITER) {
        uint64_t nonce = start_nonce + tid;
        uint8_t mining_input[84];

        // Copy header
        for (int i = 0; i < 76; i++) mining_input[i] = header[i];

        // Add nonce (little-endian)
        for (int i = 0; i < 8; ++i)
            mining_input[76 + i] = (nonce >> (8 * i)) & 0xFF;

        // First Blake2b hash
        uint8_t hash1[32];
        blake2b_cuda(hash1, mining_input, 84);

        uint64_t tmp;
        ((uint32_t*)(&tmp))[0] = cuda_swab32(((uint32_t*)&nonce)[1]);
        ((uint32_t*)(&tmp))[1] = cuda_swab32(((uint32_t*)&nonce)[0]);
        B2B_IV(aux);
        B2B_IV(aux + 8);
        aux[0] = ivals[0];
        ((uint64_t *)(aux))[12] ^= 40;
        ((uint64_t *)(aux))[13] ^= 0;
        ((uint64_t *)(aux))[14] = ~((uint64_t *)(aux))[14];
        ((uint64_t *)(aux))[16] = ((uint64_t *)hash1)[0];
        ((uint64_t *)(aux))[17] = ((uint64_t *)hash1)[1];
        ((uint64_t *)(aux))[18] = ((uint64_t *)hash1)[2];
        ((uint64_t *)(aux))[19] = ((uint64_t *)hash1)[3];
        ((uint64_t *)(aux))[20] = tmp;
        for (int i = 21; i < 32; ++i) aux[i] = 0;
        devB2B_MIX(aux, aux + 16);

        uint64_t hsh;
        #pragma unroll
        for (j = 0; j < NUM_SIZE_32; j += 2) {
            hsh = ivals[j >> 1];
            hsh ^= ((uint64_t *)(aux))[j >> 1] ^ ((uint64_t *)(aux))[8 + (j >> 1)];
            ((uint32_t*)r)[j] =  ((uint32_t*)(&hsh))[0];
            ((uint32_t*)r)[j + 1] = ((uint32_t*)(&hsh))[1];
        }

        uint32_t n_len = AUTOLYKOS2_M;
        for (int k = 0; k < K_LEN; k++) {
            uint32_t val;
            int byte_idx = (k / 4) * 4;
            if (byte_idx + 3 < 32)
                val = ((uint32_t*)r)[byte_idx / 4];
            else {
                byte_idx = byte_idx % 32;
                val = ((uint32_t*)r)[byte_idx / 4];
            }
            int sub_idx = k % 4;
            uint32_t final_val;
            if (sub_idx == 0) final_val = val;
            else if (sub_idx == 1) final_val = (val << 8) | (val >> 24);
            else if (sub_idx == 2) final_val = (val << 16) | (val >> 16);
            else final_val = (val << 24) | (val >> 8);
            ind[k] = final_val % n_len;
        }

        uint32_t current_sum[NUM_SIZE_32 + 1] = {0};
        for (int k = 0; k < K_LEN; ++k) {
            uint32_t dataset_element = dataset[ind[k]];
            uint64_t temp_sum = (uint64_t)current_sum[0] + dataset_element;
            current_sum[0] = (uint32_t)temp_sum;
            uint32_t carry = temp_sum >> 32;
            for (int i = 1; i < NUM_SIZE_32 + 1; ++i) {
                temp_sum = (uint64_t)current_sum[i] + carry;
                current_sum[i] = (uint32_t)temp_sum;
                carry = temp_sum >> 32;
            }
        }
        uint8_t sum_bytes[32];
        for(int i = 0; i < NUM_SIZE_32; ++i) {
            sum_bytes[i*4] = current_sum[i] & 0xFF;
            sum_bytes[i*4 + 1] = (current_sum[i] >> 8) & 0xFF;
            sum_bytes[i*4 + 2] = (current_sum[i] >> 16) & 0xFF;
            sum_bytes[i*4 + 3] = (current_sum[i] >> 24) & 0xFF;
        }
        uint8_t final_input[40];
        for (int i = 0; i < 32; i++) final_input[i] = hash1[i];
        for (int i = 0; i < 8; i++) final_input[32 + i] = sum_bytes[i];
        uint8_t final_hash[32];
        blake2b_cuda(final_hash, final_input, 40);

        // Compare final_hash with bound_ using 4x uint64_t little-endian words
        bool meets_target = false;
        uint64_t* hash64 = (uint64_t*)final_hash;
        uint64_t* bound64 = (uint64_t*)bound_;
        for (int i = 3; i >= 0; --i) {
            if (hash64[i] < bound64[i]) { meets_target = true; break; }
            if (hash64[i] > bound64[i]) { break; }
        }
        if (meets_target) {
            if (atomicCAS((int*)d_found_flag_param, 0, 1) == 0) {
                *d_found_nonce_param = nonce;
                for (int i = 0; i < 32; ++i) d_found_hash_param[i] = final_hash[i];
            }
        }
    }
}

__global__ void generate_dataset_kernel(uint32_t* dataset, const uint8_t* seed, uint32_t start_idx, uint32_t count) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= count) return;
    uint32_t global_idx = start_idx + idx;
    uint8_t input[36];
    for (int i = 0; i < 32; i++) input[i] = seed[i];
    input[32] = global_idx & 0xFF;
    input[33] = (global_idx >> 8) & 0xFF;
    input[34] = (global_idx >> 16) & 0xFF;
    input[35] = (global_idx >> 24) & 0xFF;
    uint8_t hash[32];
    blake2b_cuda(hash, input, 36);
    dataset[global_idx] =
        ((uint32_t)hash[0]) |
        ((uint32_t)hash[1] << 8) |
        ((uint32_t)hash[2] << 16) |
        ((uint32_t)hash[3] << 24);
}

static uint32_t* d_dataset = nullptr;
static uint8_t* d_header = nullptr;
static uint64_t* d_found_nonce = nullptr;
static uint8_t* d_found_hash = nullptr;
static bool* d_found_flag = nullptr;
static uint8_t* d_target_boundary = nullptr;
static uint32_t* h_dataset = nullptr;
static bool miner_initialized = false;

#define CUDA_CHECK_INIT(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            return false; \
        } \
    } while(0)

bool autolykos2_cuda_init(int device_id) {
    if (miner_initialized) return true;
    CUDA_CHECK_INIT(hipSetDevice(device_id));
    size_t dataset_size = AUTOLYKOS2_M * sizeof(uint32_t);
    CUDA_CHECK_INIT(hipMalloc(&d_dataset, dataset_size));
    CUDA_CHECK_INIT(hipMalloc(&d_header, 76));
    CUDA_CHECK_INIT(hipMalloc(&d_found_nonce, sizeof(uint64_t)));
    CUDA_CHECK_INIT(hipMalloc(&d_found_hash, 32));
    CUDA_CHECK_INIT(hipMalloc(&d_found_flag, sizeof(bool)));
    CUDA_CHECK_INIT(hipMalloc(&d_target_boundary, 32));
    h_dataset = (uint32_t*)malloc(dataset_size);
    if (!h_dataset) {
        fprintf(stderr, "Failed to allocate host dataset memory\n");
        return false;
    }
    miner_initialized = true;
    return true;
}

bool autolykos2_cuda_generate_dataset(const uint8_t* seed) {
    if (!miner_initialized) {
        fprintf(stderr, "Miner not initialized\n");
        return false;
    }
    uint8_t* d_temp_seed = nullptr;
    CUDA_CHECK_INIT(hipMalloc(&d_temp_seed, 32));
    CUDA_CHECK_INIT(hipMemcpy(d_temp_seed, seed, 32, hipMemcpyHostToDevice));
    const uint32_t chunk_size = 1024 * 1024;
    const uint32_t total_elements = AUTOLYKOS2_M;
    for (uint32_t start = 0; start < total_elements; start += chunk_size) {
        uint32_t count = (chunk_size < total_elements - start) ? chunk_size : (total_elements - start);
        dim3 block(BLOCK_SIZE);
        dim3 grid((count + BLOCK_SIZE - 1) / BLOCK_SIZE);
        generate_dataset_kernel<<<grid, block>>>(d_dataset, d_temp_seed, start, count);
        CUDA_CHECK_INIT(hipGetLastError());
        CUDA_CHECK_INIT(hipDeviceSynchronize());
        if (start % (chunk_size * 10) == 0) {
            printf("Dataset generation: %.2f%%\n", 100.0f * (start + count) / total_elements);
        }
    }
    CUDA_CHECK_INIT(hipFree(d_temp_seed));
    printf("Dataset generation completed\n");
    return true;
}

bool autolykos2_cuda_mine(
    const uint8_t* header,
    uint64_t start_nonce,
    uint32_t nonce_count,
    uint32_t target_hi,
    const uint8_t* target_boundary,
    uint64_t* found_nonce,
    uint8_t* found_hash,
    bool* found
) {
    if (!miner_initialized) {
        fprintf(stderr, "Miner not initialized\n");
        return false;
    }
    CUDA_CHECK_INIT(hipMemcpy(d_header, header, 76, hipMemcpyHostToDevice));
    cpyBSymbol(target_boundary);

    bool host_found = false;
    CUDA_CHECK_INIT(hipMemcpy(d_found_flag, &host_found, sizeof(bool), hipMemcpyHostToDevice));
    dim3 block(BLOCK_SIZE);
    dim3 grid((nonce_count + BLOCK_SIZE - 1) / BLOCK_SIZE);

    autolykos2_mining_kernel<<<grid, block>>>(
        d_dataset,
        d_header,
        start_nonce,
        target_hi,
        d_found_nonce,
        d_found_hash,
        d_found_flag
    );
    CUDA_CHECK_INIT(hipGetLastError());
    CUDA_CHECK_INIT(hipDeviceSynchronize());

    CUDA_CHECK_INIT(hipMemcpy(&host_found, d_found_flag, sizeof(bool), hipMemcpyDeviceToHost));
    *found = host_found;

    if (host_found) {
        CUDA_CHECK_INIT(hipMemcpy(found_nonce, d_found_nonce, sizeof(uint64_t), hipMemcpyDeviceToHost));
        CUDA_CHECK_INIT(hipMemcpy(found_hash, d_found_hash, 32, hipMemcpyDeviceToHost));
    }
    return true;
}

void autolykos2_cuda_cleanup() {
    if (!miner_initialized) return;
    if (d_dataset) hipFree(d_dataset);
    if (d_header) hipFree(d_header);
    if (d_found_nonce) hipFree(d_found_nonce);
    if (d_found_hash) hipFree(d_found_hash);
    if (d_found_flag) hipFree(d_found_flag);
    if (d_target_boundary) hipFree(d_target_boundary);
    if (h_dataset) { free(h_dataset); h_dataset = nullptr; }
    d_dataset = nullptr; d_header = nullptr; d_found_nonce = nullptr;
    d_found_flag = nullptr; d_target_boundary = nullptr;
    miner_initialized = false;
}

uint64_t autolykos2_cuda_get_hashrate() {
    return GRID_SIZE * BLOCK_SIZE * 1000;
}
bool autolykos2_cuda_is_initialized() { return miner_initialized; }

bool launchMiningKernel(
    const uint8_t* header,
    const uint8_t* target,
    uint64_t nonceStart,
    uint64_t nonceRange,
    uint64_t& foundNonce,
    uint8_t* foundHash
) {
    if (!miner_initialized) {
        fprintf(stderr, "Miner not initialized\n");
        return false;
    }
    uint64_t found_nonce_64;
    bool found = false;
    bool success = autolykos2_cuda_mine(
        header,
        nonceStart,
        (uint32_t)nonceRange,
        0,
        target,
        &found_nonce_64,
        foundHash,
        &found
    );
    if (success && found) {
        foundNonce = found_nonce_64;
        return true;
    }
    return false;
}
