#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <inttypes.h>     // <-- Add this for PRIu64
#include "blake2b_cuda.cuh"
#include "utils.h"        // <-- Use the shared inline here!

// REMOVE isHashLessThanTarget from here!
// It's now in utils.h

__global__ void mine_kernel(
    const uint8_t* header,
    const uint8_t* target,
    uint64_t nonce_start,
    uint64_t nonce_range,
    uint64_t* found_nonce,
    int* found_flag,
    uint8_t* output_hash
) {
    uint64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t total_threads = gridDim.x * blockDim.x;
    uint64_t nonce = nonce_start + thread_id;

    while (nonce < nonce_start + nonce_range) {
        if (*found_flag) return;

        uint8_t input[40];
        for (int i = 0; i < 32; i++) input[i] = header[i];
        for (int i = 0; i < 8; i++) input[32 + i] = (nonce >> (8 * i)) & 0xFF;

        uint8_t hash[32];
        blake2b_gpu(hash, input, 40);

        if (isHashLessThanTarget(hash, target)) {
            if (atomicExch(found_flag, 1) == 0) {
                *found_nonce = nonce;
                for (int i = 0; i < 32; i++) output_hash[i] = hash[i];
                // Use PRIu64 for portable printf of uint64_t
                printf("[GPU] Valid nonce found: %" PRIu64 "\n", (uint64_t)nonce);
            }
            return;
        }

        nonce += total_threads;
    }
}

