#include "hip/hip_runtime.h"
#include "autolykos2_cuda_miner.h"
#include "blake2b_cuda.cuh"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <string.h>
#include <stdint.h>

#define AUTOLYKOS2_N 26
#define AUTOLYKOS2_K 32
#define AUTOLYKOS2_M (1 << AUTOLYKOS2_N)
#define BLOCK_SIZE 256
#define GRID_SIZE 1024

#define NONCES_PER_ITER (BLOCK_SIZE * GRID_SIZE)
#define THREADS_PER_ITER BLOCK_SIZE
#define NUM_SIZE_32 8
#define K_LEN 64
#define BUF_SIZE_8 128
#define MAX_SOLS 10

#define B2B_IV(h) \
    do { \
        ((uint64_t *)(h))[0] = 0x6A09E667F2BDC928ULL; \
        ((uint64_t *)(h))[1] = 0xBB67AE8584CAA73BULL; \
        ((uint64_t *)(h))[2] = 0x3C6EF372FE94F82BULL; \
        ((uint64_t *)(h))[3] = 0xA54FF53A5F1D36F1ULL; \
        ((uint64_t *)(h))[4] = 0x510E527FADE682D1ULL; \
        ((uint64_t *)(h))[5] = 0x9B05688C2B3E6C1FULL; \
        ((uint64_t *)(h))[6] = 0x1F83D9ABFB41BD6BULL; \
        ((uint64_t *)(h))[7] = 0x5BE0CD19137E2179ULL; \
    } while(0)

__device__ __forceinline__ uint32_t ld_gbl_cs(const  uint32_t * __restrict__ p) {
    uint32_t v;
    asm("ld.global.cs.u32 %0, [%1];" : "=r"(v) : "l"(p));
    return v;
}
__device__ __forceinline__ uint4 ld_gbl_cs_v4(const  uint4 * __restrict__ p) {
    uint4 v;
    asm("ld.global.cs.v4.u32 {%0, %1, %2, %3}, [%4];" : "=r"(v.x), "=r"(v.y), "=r"(v.z), "=r"(v.w) : "l"(p));
    return v;
}
__device__ __forceinline__ uint32_t cuda_swab32(uint32_t x) {
    return __byte_perm(x, x, 0x0123);
}
__device__ __forceinline__ uint64_t devectorize(uint2 x) {
    uint64_t result;
    asm("mov.b64 %0,{%1,%2}; \n\t"
        : "=l"(result) : "r"(x.x), "r"(x.y));
    return result;
}
__device__ __forceinline__ uint2 vectorize(const uint64_t x) {
    uint2 result;
    asm("mov.b64 {%0,%1},%2; \n\t"
        : "=r"(result.x), "=r"(result.y) : "l"(x));
    return result;
}
__device__ __forceinline__
uint64_t devROTR64(uint64_t b, int offset) {
    uint2 a;
    uint2 result;
    a = vectorize(b);

    if (offset < 32) {
        asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.x), "r"(a.y), "r"(offset));
        asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.y), "r"(a.x), "r"(offset));
    }
    else {
        asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.y), "r"(a.x), "r"(offset));
        asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.x), "r"(a.y), "r"(offset));
    }
    return devectorize(result);
}

__device__ __forceinline__
void devB2B_G(uint64_t* v, int a, int b, int c, int d, uint64_t x, uint64_t y) {
    ((uint64_t *)(v))[a] += ((uint64_t *)(v))[b] + x;
    ((uint64_t *)(v))[d]
        = devROTR64(((uint64_t *)(v))[d] ^ ((uint64_t *)(v))[a], 32);
    ((uint64_t *)(v))[c] += ((uint64_t *)(v))[d];
    ((uint64_t *)(v))[b]
        = devROTR64(((uint64_t *)(v))[b] ^ ((uint64_t *)(v))[c], 24);
    ((uint64_t *)(v))[a] += ((uint64_t *)(v))[b] + y;
    ((uint64_t *)(v))[d]
        = devROTR64(((uint64_t *)(v))[d] ^ ((uint64_t *)(v))[a], 16);
    ((uint64_t *)(v))[c] += ((uint64_t *)(v))[d];
    ((uint64_t *)(v))[b]
        = devROTR64(((uint64_t *)(v))[b] ^ ((uint64_t *)(v))[c], 63);
}

__device__ __forceinline__
void devB2B_MIX(uint64_t* v, uint64_t* m) {
    // ... (same as your posted code, omitted for brevity, leave unchanged)
    // Keep your MIX implementation as-is unless you know it's wrong.
    // Otherwise this answer is too long for ChatGPT's limits.
    // (Just keep your existing body here.)
}

const __constant__ uint64_t ivals[8] = {
    0x6A09E667F2BDC928,
    0xBB67AE8584CAA73B,
    0x3C6EF372FE94F82B,
    0xA54FF53A5F1D36F1,
    0x510E527FADE682D1,
    0x9B05688C2B3E6C1F,
    0x1F83D9ABFB41BD6B,
    0x5BE0CD19137E2179
};

__constant__ uint8_t bound_[32];

// Function to copy the target boundary to constant memory
void cpyBSymbol(const uint8_t *bound) {
    hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(bound_), bound, 32);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error in cpyBSymbol: %s\n", hipGetErrorString(err));
    }
}

// --- KERNELS ---

__global__ void autolykos2_mining_kernel(
    const uint32_t* dataset,
    const uint8_t* header,
    uint64_t start_nonce,
    uint32_t target_hi, // unused
    uint64_t* d_found_nonce_param,
    bool* d_found_flag_param
) {
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t aux[32] = { 0 };
    uint32_t ind[K_LEN] = { 0 };
    uint32_t r[NUM_SIZE_32 + 1] = { 0 };
    uint8_t j = 0;

    if (tid < NONCES_PER_ITER) {
        uint64_t nonce = start_nonce + tid;
        uint8_t mining_input[84];

        // Copy header
        for (int i = 0; i < 76; i++) mining_input[i] = header[i];

        // Add nonce (little-endian)
        for (int i = 0; i < 8; ++i)
            mining_input[76 + i] = (nonce >> (8 * i)) & 0xFF;

        // First Blake2b hash
        uint8_t hash1[32];
        blake2b_cuda(hash1, mining_input, 84);

        uint64_t tmp;
        ((uint32_t*)(&tmp))[0] = cuda_swab32(((uint32_t*)&nonce)[1]);
        ((uint32_t*)(&tmp))[1] = cuda_swab32(((uint32_t*)&nonce)[0]);
        B2B_IV(aux);
        B2B_IV(aux + 8);
        aux[0] = ivals[0];
        ((uint64_t *)(aux))[12] ^= 40;
        ((uint64_t *)(aux))[13] ^= 0;
        ((uint64_t *)(aux))[14] = ~((uint64_t *)(aux))[14];
        ((uint64_t *)(aux))[16] = ((uint64_t *)hash1)[0];
        ((uint64_t *)(aux))[17] = ((uint64_t *)hash1)[1];
        ((uint64_t *)(aux))[18] = ((uint64_t *)hash1)[2];
        ((uint64_t *)(aux))[19] = ((uint64_t *)hash1)[3];
        ((uint64_t *)(aux))[20] = tmp;
        for (int i = 21; i < 32; ++i) aux[i] = 0;
        devB2B_MIX(aux, aux + 16);

        uint64_t hsh;
        #pragma unroll
        for (j = 0; j < NUM_SIZE_32; j += 2) {
            hsh = ivals[j >> 1];
            hsh ^= ((uint64_t *)(aux))[j >> 1] ^ ((uint64_t *)(aux))[8 + (j >> 1)];
            ((uint32_t*)r)[j] =  ((uint32_t*)(&hsh))[0];
            ((uint32_t*)r)[j + 1] = ((uint32_t*)(&hsh))[1];
        }

        // Index generation
        uint32_t n_len = AUTOLYKOS2_M;
        for (int k = 0; k < K_LEN; k++) {
            uint32_t val;
            int byte_idx = (k / 4) * 4;
            if (byte_idx + 3 < 32)
                val = ((uint32_t*)r)[byte_idx / 4];
            else {
                byte_idx = byte_idx % 32;
                val = ((uint32_t*)r)[byte_idx / 4];
            }
            int sub_idx = k % 4;
            uint32_t final_val;
            if (sub_idx == 0) final_val = val;
            else if (sub_idx == 1) final_val = (val << 8) | (val >> 24);
            else if (sub_idx == 2) final_val = (val << 16) | (val >> 16);
            else final_val = (val << 24) | (val >> 8);
            ind[k] = final_val % n_len;
        }

        uint32_t current_sum[NUM_SIZE_32 + 1] = {0};
        for (int k = 0; k < K_LEN; ++k) {
            uint32_t dataset_element = dataset[ind[k]];
            uint64_t temp_sum = (uint64_t)current_sum[0] + dataset_element;
            current_sum[0] = (uint32_t)temp_sum;
            uint32_t carry = temp_sum >> 32;
            for (int i = 1; i < NUM_SIZE_32 + 1; ++i) {
                temp_sum = (uint64_t)current_sum[i] + carry;
                current_sum[i] = (uint32_t)temp_sum;
                carry = temp_sum >> 32;
            }
        }
        uint8_t sum_bytes[32];
        for(int i = 0; i < NUM_SIZE_32; ++i) {
            sum_bytes[i*4] = current_sum[i] & 0xFF;
            sum_bytes[i*4 + 1] = (current_sum[i] >> 8) & 0xFF;
            sum_bytes[i*4 + 2] = (current_sum[i] >> 16) & 0xFF;
            sum_bytes[i*4 + 3] = (current_sum[i] >> 24) & 0xFF;
        }
        uint8_t final_input[40];
        for (int i = 0; i < 32; i++) final_input[i] = hash1[i];
        for (int i = 0; i < 8; i++) final_input[32 + i] = sum_bytes[i];
        uint8_t final_hash[32];
        blake2b_cuda(final_hash, final_input, 40);

        // Compare final_hash with bound_
        bool meets_target = false;
        for (int i = 31; i >= 0; --i) {
            if (final_hash[i] < bound_[i]) { meets_target = true; break; }
            if (final_hash[i] > bound_[i]) { break; }
        }
        if (meets_target) {
            if (atomicCAS((int*)d_found_flag_param, 0, 1) == 0) {
                *d_found_nonce_param = nonce;
                // Could save hash if needed
            }
        }
    }
}

// CUDA error checking macro
#define CUDA_CHECK_INIT(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            return false; \
        } \
    } while(0)

static uint32_t* d_dataset = nullptr;
static uint8_t* d_header = nullptr;
static uint64_t* d_found_nonce = nullptr;
static bool* d_found_flag = nullptr;
static uint8_t* d_target_boundary = nullptr;
static uint32_t* h_dataset = nullptr;
static bool miner_initialized = false;

// Generate Autolykos2 dataset on GPU
__global__ void generate_dataset_kernel(uint32_t* dataset, const uint8_t* seed, uint32_t start_idx, uint32_t count) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= count) return;
    uint32_t global_idx = start_idx + idx;
    uint8_t input[36];
    for (int i = 0; i < 32; i++) input[i] = seed[i];
    input[32] = global_idx & 0xFF;
    input[33] = (global_idx >> 8) & 0xFF;
    input[34] = (global_idx >> 16) & 0xFF;
    input[35] = (global_idx >> 24) & 0xFF;
    uint8_t hash[32];
    blake2b_cuda(hash, input, 36);
    dataset[global_idx] =
        ((uint32_t)hash[0]) |
        ((uint32_t)hash[1] << 8) |
        ((uint32_t)hash[2] << 16) |
        ((uint32_t)hash[3] << 24);
}

bool autolykos2_cuda_init(int device_id) {
    if (miner_initialized) return true;
    CUDA_CHECK_INIT(hipSetDevice(device_id));
    size_t dataset_size = AUTOLYKOS2_M * sizeof(uint32_t);
    CUDA_CHECK_INIT(hipMalloc(&d_dataset, dataset_size));
    CUDA_CHECK_INIT(hipMalloc(&d_header, 76));
    CUDA_CHECK_INIT(hipMalloc(&d_found_nonce, sizeof(uint64_t)));
    CUDA_CHECK_INIT(hipMalloc(&d_found_flag, sizeof(bool)));
    CUDA_CHECK_INIT(hipMalloc(&d_target_boundary, 32));
    h_dataset = (uint32_t*)malloc(dataset_size);
    if (!h_dataset) {
        fprintf(stderr, "Failed to allocate host dataset memory\n");
        return false;
    }
    miner_initialized = true;
    return true;
}

bool autolykos2_cuda_generate_dataset(const uint8_t* seed) {
    if (!miner_initialized) {
        fprintf(stderr, "Miner not initialized\n");
        return false;
    }
    uint8_t* d_temp_seed = nullptr;
    CUDA_CHECK_INIT(hipMalloc(&d_temp_seed, 32));
    CUDA_CHECK_INIT(hipMemcpy(d_temp_seed, seed, 32, hipMemcpyHostToDevice));
    const uint32_t chunk_size = 1024 * 1024;
    const uint32_t total_elements = AUTOLYKOS2_M;
    for (uint32_t start = 0; start < total_elements; start += chunk_size) {
        uint32_t count = (chunk_size < total_elements - start) ? chunk_size : (total_elements - start);
        dim3 block(BLOCK_SIZE);
        dim3 grid((count + BLOCK_SIZE - 1) / BLOCK_SIZE);
        generate_dataset_kernel<<<grid, block>>>(d_dataset, d_temp_seed, start, count);
        CUDA_CHECK_INIT(hipGetLastError());
        CUDA_CHECK_INIT(hipDeviceSynchronize());
        if (start % (chunk_size * 10) == 0) {
            printf("Dataset generation: %.2f%%\n", 100.0f * (start + count) / total_elements);
        }
    }
    CUDA_CHECK_INIT(hipFree(d_temp_seed));
    printf("Dataset generation completed\n");
    return true;
}

bool autolykos2_cuda_mine(
    const uint8_t* header,
    uint64_t start_nonce,
    uint32_t nonce_count,
    uint32_t target_hi,
    const uint8_t* target_boundary,
    uint64_t* found_nonce,
    bool* found
) {
    if (!miner_initialized) {
        fprintf(stderr, "Miner not initialized\n");
        return false;
    }
    CUDA_CHECK_INIT(hipMemcpy(d_header, header, 76, hipMemcpyHostToDevice));

    // --- endian swap for pool targets ---
    uint8_t bound_be[32];
    for (int i = 0; i < 32; ++i) bound_be[i] = target_boundary[31 - i];
    cpyBSymbol(bound_be);

    bool host_found = false;
    CUDA_CHECK_INIT(hipMemcpy(d_found_flag, &host_found, sizeof(bool), hipMemcpyHostToDevice));
    dim3 block(BLOCK_SIZE);
    dim3 grid((nonce_count + BLOCK_SIZE - 1) / BLOCK_SIZE);

    autolykos2_mining_kernel<<<grid, block>>>(
        d_dataset,
        d_header,
        start_nonce,
        target_hi,
        d_found_nonce,
        d_found_flag
    );
    CUDA_CHECK_INIT(hipGetLastError());
    CUDA_CHECK_INIT(hipDeviceSynchronize());

    CUDA_CHECK_INIT(hipMemcpy(&host_found, d_found_flag, sizeof(bool), hipMemcpyDeviceToHost));
    *found = host_found;

    if (host_found) {
        CUDA_CHECK_INIT(hipMemcpy(found_nonce, d_found_nonce, sizeof(uint64_t), hipMemcpyDeviceToHost));
    }
    return true;
}

void autolykos2_cuda_cleanup() {
    if (!miner_initialized) return;
    if (d_dataset) hipFree(d_dataset);
    if (d_header) hipFree(d_header);
    if (d_found_nonce) hipFree(d_found_nonce);
    if (d_found_flag) hipFree(d_found_flag);
    if (d_target_boundary) hipFree(d_target_boundary);
    if (h_dataset) { free(h_dataset); h_dataset = nullptr; }
    d_dataset = nullptr; d_header = nullptr; d_found_nonce = nullptr;
    d_found_flag = nullptr; d_target_boundary = nullptr;
    miner_initialized = false;
}

uint64_t autolykos2_cuda_get_hashrate() {
    return GRID_SIZE * BLOCK_SIZE * 1000;
}
bool autolykos2_cuda_is_initialized() { return miner_initialized; }

bool launchMiningKernel(
    const uint8_t* header,
    const uint8_t* target,
    uint64_t nonceStart,
    uint64_t nonceRange,
    uint64_t& foundNonce,
    uint8_t* foundHash
) {
    if (!miner_initialized) {
        fprintf(stderr, "Miner not initialized\n");
        return false;
    }
    uint64_t found_nonce_64;
    bool found = false;
    bool success = autolykos2_cuda_mine(
        header,
        nonceStart,
        (uint32_t)nonceRange,
        0,
        target,
        &found_nonce_64,
        &found
    );
    if (success && found) {
        foundNonce = found_nonce_64;
        // foundHash - left for extension if needed
        return true;
    }
    return false;
}
